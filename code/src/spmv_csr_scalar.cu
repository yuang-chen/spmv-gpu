#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "mmio.h"

#define BlockDim 1024
#define ITER 3

template <typename T>
__global__ void spmv_csr_scalar_kernel(T * d_val,T * d_vector,int * d_cols,int * d_ptr,int N, T * d_out)
{
    	int tid = blockIdx.x * blockDim.x + threadIdx.x;

    	for (int i = tid; i < N; i += blockDim.x * gridDim.x)
    	{
        	T t = 0;
        	int start = d_ptr[i];
        	int end = d_ptr[i+1];
		// One thread handles all elements of the row assigned to it
        	for (int j = start; j < end; j++)
        	{
            		int col = d_cols[j];
            		t += d_val[j] * d_vector[col];
        	}
        	d_out[i] = t;
    	}
}

template <typename T>
void spmv_csr_scalar(MatrixInfo<T> * mat,T *vector,T *out) 
{
    	T *d_vector,*d_val, *d_out;
    	int *d_cols, *d_ptr;
    	float time_taken;
    	double gflop = 2 * (double) mat->nz / 1e9;
    	float milliseconds = 0;
    	hipEvent_t start, stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);

	// Allocate memory on device
    	hipMalloc(&d_vector,mat->N*sizeof(T));
    	hipMalloc(&d_val,mat->nz*sizeof(T));
    	hipMalloc(&d_out,mat->M*sizeof(T));
    	hipMalloc(&d_cols,mat->nz*sizeof(int));
    	hipMalloc(&d_ptr,(mat->M+1)*sizeof(int));

	// Copy from host memory to device memory
    	hipMemcpy(d_vector,vector,mat->N*sizeof(T),hipMemcpyHostToDevice);
    	hipMemcpy(d_val,mat->val,mat->nz*sizeof(T),hipMemcpyHostToDevice);
    	hipMemcpy(d_cols,mat->cIndex,mat->nz*sizeof(int),hipMemcpyHostToDevice);
    	hipMemcpy(d_ptr,mat->rIndex,(mat->M+1)*sizeof(int),hipMemcpyHostToDevice);
    	hipMemset(d_out, 0, mat->M*sizeof(T));

	// Run the kernel and time it
    	hipEventRecord(start);
    	for (int i = 0; i < ITER; i++)
 		spmv_csr_scalar_kernel<T><<<ceil(mat->M/(float)BlockDim),BlockDim>>>(d_val,d_vector,d_cols,d_ptr,mat->M,d_out);
    	hipEventRecord(stop);
    	hipEventSynchronize(stop);
    	hipEventElapsedTime(&milliseconds, start, stop);

	// Copy from device memory to host memory 
    	hipMemcpy(out, d_out, mat->M*sizeof(T), hipMemcpyDeviceToHost);

	// Free device memory
    	hipFree(d_vector);
    	hipFree(d_val);
    	hipFree(d_cols);
    	hipFree(d_ptr); 
    	hipFree(d_out);

	// Calculate and print out GFLOPs and GB/s 
	double gbs = ((mat->N * sizeof(T)) + (mat->nz*sizeof(T)) + (mat->M*sizeof(int)) + (mat->nz*sizeof(int)) + (mat->M*sizeof(T))) / (milliseconds/ITER) / 1e6;
    	time_taken = (milliseconds/ITER)/1000.0; 
    	printf("Average time taken for %s is %f\n", "SpMV by GPU CSR Scalar Algorithm",time_taken);
    	printf("Average GFLOP/s is %lf\n",gflop/time_taken);
	printf("Average GB/s is %lf\n\n",gbs);
}
